#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


void jacobi(float* A, float* b, int N, int iter, float** x)
{
   int i, j, k;
   float t;
   float *y;

   *x = (float*) malloc(N*sizeof(float));
   y = (float*) malloc(N*sizeof(float));
   for(i = 0; i < N; i++)
   {
      (*x)[i] = 0.0; // Initial Guess
   }

   for(k = 0; k < iter; k++)
   {
      for(i = 0; i < N; i++)
      {
         t = 0;
         for(j = 0; j < N; j++)
         {
            if(i != j)
            {
               t += ((A[ ((N*i)+j) ]) * ((*x)[j]));
            }
         }
         y[i] = ((b[i]) - t)/(A[ ((N*i)+i) ]);
         //printf("k %02d i %02d t %f y %f\n", k, i, t, y[i]);
      }

      for(i = 0; i < N; i++)
      {
         (*x)[i] = y[i];
      }
   }
   free(y);
}




__global__
void iloop(float* A, float* b, int N, float* x, float* y, int offset, int streamsize)
{
	float t;

	int index = offset + (blockIdx.x * blockDim.x) +  threadIdx.x;
	int stride = (gridDim.x * blockDim.x);

	//printf("<<< %d, %d, %d >>>\n", index, (offset+streamsize), stride);

	for(int i = index; i < (offset + streamsize); i += stride)
	{
		t = 0.0;
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				t = t + (( A[ ((N*i)+j) ] ) * x[j]);
			}
		}
		y[i] = ((b[i] - t)/(A[ ((N*i)+i) ]));
	}
}

__global__
void add(float* a, float* b, int N)
{
	for(int i = 0; i < N; i++)
	{
		b[i] = a[i] + b[i];
	}
}




int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;
	int nstreams;
	int streamsize;
	int devID;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc >= 2) fname = argv[1];
   else fname = "../inputs/8.txt";

	//if(argc >= 3) devID = atoi(argv[2]);
	//else devID = 1;

   //cudaDeviceProp prop;
	//if( cudaGetDeviceProperties(&prop, devID) != cudaSuccess) printf("CUDA : Error with Device Properties\n");
	//else printf("Device : %d\n", prop.id);
	//if( cudaSetDevice(devID) != cudaSuccess ) printf("CUDA : Error with Device ID\n");
	
	
	readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));

	memset(x, 0.0, N*sizeof(float));
	memset(y, 0.0, N*sizeof(float));
	//for(i = 0; i < N; i++)
	//{
	//	x[i] = 0.0;
	//	y[i] = 0.0;
	//}

	if( hipMalloc(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	if( hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_b, b, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_y, y, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");

	nstreams = 2;
	streamsize = N/nstreams;
	blocksize = (int) fmin(256, streamsize);
	numblocks = streamsize/blocksize;
	//numblocks = (N+blocksize-1)/blocksize;
	printf("CUDA : Streams %d, Grid Size %d, Block size %d\n", nstreams, numblocks, blocksize);
 
	hipStream_t streams[nstreams];
   for(i = 0; i < nstreams; i++)
	{
		if( hipStreamCreate(&streams[i]) != hipSuccess ) printf("CUDA : Error with Stream Creation\n");
	}
	
	hipEventRecord(start);
   
	//if( cudaMemcpy(d_x, x, N*sizeof(float)  , cudaMemcpyHostToDevice) != cudaSuccess ) printf("CUDA : memory copy error!\n");
	for(k = 0; k < iter; k++)
	{
		for(i = 0; i < nstreams; i++)
		{
			int offset = i*streamsize;
			if( hipMemcpyAsync(&d_x[offset], &d_y[offset], streamsize*sizeof(float), hipMemcpyDeviceToDevice, streams[i])!= hipSuccess ) printf("CUDA : memory copy error!\n");
		//}
		//for(i = 0; i < nstreams; i++)
		//{
		//	int offset = i*streamsize;
			iloop<<< numblocks, blocksize, 0, streams[i] >>>(d_A, d_b, N, d_x, d_y, offset, streamsize); // kernel launch on GPU
		}
	}
   if( hipMemcpy(x, d_y, N*sizeof(float)  , hipMemcpyDeviceToHost) != hipSuccess ) printf("CUDA : memory copy error!\n");
	
   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	for(i = 0; i < nstreams; i++)
	{
	   if( hipStreamDestroy(streams[i]) != hipSuccess ) printf("CUDA : Error with Stream Deletion\n");
	}

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
