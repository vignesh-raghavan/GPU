#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


void jacobi(float* A, float* b, int N, int iter, float** x)
{
   int i, j, k;
   float t;
   float *y;

   *x = (float*) malloc(N*sizeof(float));
   y = (float*) malloc(N*sizeof(float));
   for(i = 0; i < N; i++)
   {
      (*x)[i] = 0.0; // Initial Guess
   }

   for(k = 0; k < iter; k++)
   {
      for(i = 0; i < N; i++)
      {
         t = 0;
         for(j = 0; j < N; j++)
         {
            if(i != j)
            {
               t += ((A[ ((N*i)+j) ]) * ((*x)[j]));
            }
         }
         y[i] = ((b[i]) - t)/(A[ ((N*i)+i) ]);
         //printf("k %02d i %02d t %f y %f\n", k, i, t, y[i]);
      }

      for(i = 0; i < N; i++)
      {
         (*x)[i] = y[i];
      }
   }
   free(y);
}




__global__
void iloop(float* A, float* b, int N, float* x, float* y)
{
	float t;

	int index = (blockIdx.x * blockDim.x) +  threadIdx.x;
	int stride = (gridDim.x * blockDim.x);

	//printf("<<< %d, %f, %f >>>\n", N, A[0], b[0]);

	for(int i = index; i < N; i += stride)
	{
		t = 0.0;
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				t = t + (( A[ ((N*i)+j) ] ) * x[j]);
			}
		}
		y[i] = ((b[i] - t)/(A[ ((N*i)+i) ]));
	}
}

__global__
void add(float* a, float* b, int N)
{
	for(int i = 0; i < N; i++)
	{
		b[i] = a[i] + b[i];
	}
}




int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc == 2) fname = argv[1];
   else fname = "../inputs/8.txt";

   readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));
	
	for(i = 0; i < N; i++)
	{
		x[i] = 0.0;
		y[i] = 0.0;
	}

	if( hipMalloc(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	if( hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_b, b, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_y, y, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	//for(i = 0; i < N*N; i++)
	//{
	//	d_A[i] = A[i];
	//}
	//for(i = 0; i < N; i++)
	//{
	//	d_b[i] = b[i];
	//	d_x[i] = x[i];
	//	d_y[i] = y[i];
	//}

	blocksize = 256;
	numblocks = (N+blocksize-1)/blocksize;
	printf("CUDA : Grid Size %d, Block size %d\n", numblocks, blocksize);

	hipEventRecord(start);
   
	if( hipMemcpy(d_x, x, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	for(k = 0; k < iter/2; k++)
	{
	   //if( cudaMemcpy(d_x, x, N*sizeof(float)  , cudaMemcpyHostToDevice) != cudaSuccess ) printf("CUDA : memory copy error!\n");
		iloop<<< numblocks, blocksize >>>(d_A, d_b, N, d_x, d_y); // kernel launch on GPU
	   //if( cudaMemcpy(d_x, d_y, N*sizeof(float)  , cudaMemcpyDeviceToDevice) != cudaSuccess ) printf("CUDA : memory copy error!\n");
		//cudaDeviceSynchronize();
		iloop<<< numblocks, blocksize >>>(d_A, d_b, N, d_y, d_x); // kernel launch on GPU
		//cudaDeviceSynchronize();
	}
   if( hipMemcpy(x, d_x, N*sizeof(float)  , hipMemcpyDeviceToHost) != hipSuccess ) printf("CUDA : memory copy error!\n");
	
	//add<<<1, 1>>>(d_x, d_y, N);
	//cudaDeviceSynchronize();
	//for(j = 0; j < N; j++) printf("CUDA : %f\n", d_y[j]);

   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
	//Verify : sdiff -s <output> inputs/<input>
   //for(i = 0; i < N; i++)
   //{
   //   for(j = 0; j < N; j++)
   //   {
   //      printf("%d", (int)A[i][j]);
   //      if(j < N-1) printf(" ");
   //   }
   //   printf("\n");
   //}

   //for(i = 0; i < N; i++)
   //{
   //   printf("%d\n", (int)b[i]);
   //}
   //printf("\n\n");

   //for(i = 0; i < N; i++)
   //{
   //   printf("%f\n", x[i]);
   //}
   //printf("\n\n");

	//for(i = 0; i < N; i++) x[i] = d_x[i];

   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
