#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


void jacobi(float** A, float** b, int* N, int* iter, float** x)
{
   int i, j, k;
   float t;
   float *y;

   *x = (float*) malloc(*N*sizeof(float));
   y = (float*) malloc(*N*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      (*x)[i] = 0.0; // Initial Guess
   }

   for(k = 0; k < *iter; k++)
   {
      for(i = 0; i < *N; i++)
      {
         t = 0;
         for(j = 0; j < *N; j++)
         {
            if(i != j)
            {
               t += (((*A)[ ((*N*i)+j) ]) * ((*x)[j]));
            }
         }
         y[i] = (((*b)[i]) - t)/((*A)[ ((*N*i)+i) ]);
         //printf("k %02d i %02d t %f y %f\n", k, i, t, y[i]);
      }

      for(i = 0; i < *N; i++)
      {
         (*x)[i] = y[i];
      }
   }
   //free(y);
}




void iloop(float* A, float* b, int N, float* x, float* y)
{
	float t;

	for(int i = 0; i < N; i++)
	{
		t = 0.0;
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				t = t + (( A[ ((N*i)+j) ] ) * x[j]);
			}
		}
		y[i] = ((b[i] - t)/( A[ ((N*i)+i) ]));
	}
}






int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* M;
	int k;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc == 2) fname = argv[1];
   else fname = "../inputs/8.txt";

   readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file%s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));

	for(i = 0; i < N; i++)
	{
		x[i] = 0.0;
		y[i] = 0.0;
	}

	hipEventRecord(start);
   
	//jacobi(&A, &b, &N, &iter, &x);

	for(k = 0; k < iter; k++)
	{
		iloop(A, b, N, x, y);
		for(j = 0; j < N; j++) x[j] = y[j];
	}

   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on CPU\n");

   //for(i = 0; i < N; i++)
   //{
   //   for(j = 0; j < N; j++)
   //   {
   //      printf("%d", (int)A[i][j]);
   //      if(j < N-1) printf(" ");
   //   }
   //   printf("\n");
   //}

   //for(i = 0; i < N; i++)
   //{
   //   printf("%d\n", (int)b[i]);
   //}
   //printf("\n\n");

   //for(i = 0; i < N; i++)
   //{
   //   printf("%f\n", x[i]);
   //}
   //printf("\n\n");

   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
