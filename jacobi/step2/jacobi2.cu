#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


__global__
void iloop(float* A, float* b, int N, float* x, float* y)
{
	float t;

	int index = (blockIdx.x * blockDim.x) +  threadIdx.x;
	int stride = (gridDim.x * blockDim.x);

	//printf("<<< %d, %f, %f >>>\n", N, A[0], b[0]);

	for(int i = index; i < N; i += stride)
	{
		t = 0.0;
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				t = t + (( A[ ((N*i)+j) ] ) * x[j]);
			}
		}
		y[i] = ((b[i] - t)/(A[ ((N*i)+i) ]));
	}
}



int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, M, iter, i, j;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc >= 2) fname = argv[1];
   else fname = "../inputs/8.txt";
   if(argc >= 3) M = atoi(argv[2]);
   else M = 32;

   readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));

	for(i = 0; i < N; i++)
	{
		x[i] = 0.0;
		y[i] = 0.0;
	}

	if( hipMallocManaged(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMallocManaged(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMallocManaged(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMallocManaged(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	for(i = 0; i < N*N; i++)
	{
		d_A[i] = A[i];
	}
	for(i = 0; i < N; i++)
	{
		d_b[i] = b[i];
		d_x[i] = x[i];
		d_y[i] = y[i];
	}

	blocksize = M;
	numblocks = (N+blocksize-1)/blocksize;
	printf("CUDA : Grid Size %d, Block size %d\n", numblocks, blocksize);

	hipEventRecord(start);
   
	for(k = 0; k < iter; k++)
	{
		iloop<<< numblocks, blocksize >>>(d_A, d_b, N, d_x, d_y); // kernel launch on GPU
		hipDeviceSynchronize();
		for(j = 0; j < N; j++) d_x[j] = d_y[j];
	}

	//cudaDeviceSynchronize();
	//for(j = 0; j < N; j++) printf("CUDA : %f\n", d_y[j]);

   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
	//Verify : sdiff -s <output> inputs/<input>
   //for(i = 0; i < N; i++)
   //{
   //   for(j = 0; j < N; j++)
   //   {
   //      printf("%d", (int)A[i][j]);
   //      if(j < N-1) printf(" ");
   //   }
   //   printf("\n");
   //}

   //for(i = 0; i < N; i++)
   //{
   //   printf("%d\n", (int)b[i]);
   //}
   //printf("\n\n");

   //for(i = 0; i < N; i++)
   //{
   //   printf("%f\n", x[i]);
   //}
   //printf("\n\n");
   
	for(i = 0; i < N; i++) x[i] = d_x[i];
   
	c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
