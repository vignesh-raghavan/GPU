#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


__global__
void iloop(float* A, float* b, int N, float* x, float* y)
{
	extern __shared__ float shMem[];
	float* shX = shMem;
	float* shT = (float*) &shMem[N];

	int bid = blockIdx.x;
	int bstride = gridDim.x;
	int tid = threadIdx.x;
	int tstride = blockDim.x;
	int i, j, k;
	//float t;

	//printf("<<< %d, %f, %f >>>\n", N, A[0], b[0]);
	for(k = tid; k < N; k += tstride) 
	{
		shX[k] = x[k];
	}

	__syncthreads();

	for(i = bid; i < N; i += bstride)
	{
		//t = 0.0;
		shT[tid] = 0.0;
		for(j = tid; j < N; j += tstride)
		{
			shT[tid] += A[ ((N*i)+j) ] * shX[j];
		}
		//shT[tid] = t;

		__syncthreads();

		for(k = tstride/2; k > 0; k>>=1)
		{
			if(tid < k) shT[tid] += shT[ (tid+k) ];

			__syncthreads();
		}

		if(tid == 0) y[i] = shX[i] + ( (b[i] - shT[0])/(A[ ((N*i)+i) ]) );
	}
}



int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc >= 2) fname = argv[1];
   else fname = "../inputs/8.txt";

   readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));
	
	for(i = 0; i < N; i++)
	{
		x[i] = 0.0;
		y[i] = 0.0;
	}

	if( hipMalloc(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	if( hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_b, b, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_y, y, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	//for(i = 0; i < N*N; i++)
	//{
	//	d_A[i] = A[i];
	//}
	//for(i = 0; i < N; i++)
	//{
	//	d_b[i] = b[i];
	//	d_x[i] = x[i];
	//	d_y[i] = y[i];
	//}

	blocksize = 1024;//fmin(1024, N);
	//numblocks = (N+blocksize-1)/blocksize;
	numblocks = 112; 
	printf("CUDA : Grid Size %d, Block size %d\n", numblocks, blocksize);

	hipEventRecord(start);
   
	if( hipMemcpy(d_x, x, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	for(k = 0; k < iter; k++)
	{
		iloop<<< numblocks, blocksize, ((N+blocksize)*sizeof(float)) >>>(d_A, d_b, N, d_x, d_y); // kernel launch on GPU
	   if( hipMemcpy(d_x, d_y, N*sizeof(float)  , hipMemcpyDeviceToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
		//if(k%100 == 0)printf("CUDA: %d\n", k);
	}
   if( hipMemcpy(x, d_x, N*sizeof(float)  , hipMemcpyDeviceToHost) != hipSuccess ) printf("CUDA : memory copy error!\n");
	
	//cudaDeviceSynchronize();
	//for(j = 0; j < N; j++) printf("CUDA : %f\n", d_y[j]);

   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
	//Verify : sdiff -s <output> inputs/<input>
   //for(i = 0; i < N; i++)
   //{
   //   for(j = 0; j < N; j++)
   //   {
   //      printf("%d", (int)A[i][j]);
   //      if(j < N-1) printf(" ");
   //   }
   //   printf("\n");
   //}

   //for(i = 0; i < N; i++)
   //{
   //   printf("%d\n", (int)b[i]);
   //}
   //printf("\n\n");

   //for(i = 0; i < N; i++)
   //{
   //   printf("%f\n", x[i]);
   //}
   //printf("\n\n");

	//for(i = 0; i < N; i++) x[i] = d_x[i];

   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
