#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}


__global__
void iloop(float* A, float* b, int N, float* x, float* y)
{
	extern __shared__ float X[];

	float t = 0.0;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j;

	for(j = threadIdx.x; j < N; j += blockDim.x)
	{
		X[j] = x[j];
	}

	__syncthreads();

	for(j = 0; j < N; j++)
	{
		t += (( A[ ((N*i)+j) ] ) * X[j]);
	}
	y[i] = X[i] + ( (b[i] - t)/(A[ ((N*i)+i) ]) );
}



int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j, M;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc >= 2) fname = argv[1];
   else fname = "../inputs/8.txt";
   if(argc >= 3) M = atoi(argv[2]);
   else M = 32;

   readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));
	
	for(i = 0; i < N; i++)
	{
		x[i] = 0.0;
		y[i] = 0.0;
	}

	if( hipMalloc(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	if( hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_b, b, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_y, y, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	//for(i = 0; i < N*N; i++)
	//{
	//	d_A[i] = A[i];
	//}
	//for(i = 0; i < N; i++)
	//{
	//	d_b[i] = b[i];
	//	d_x[i] = x[i];
	//	d_y[i] = y[i];
	//}

	blocksize = fmin(M, N);
	numblocks = (N+blocksize-1)/blocksize;
	printf("CUDA : Grid Size %d, Block size %d\n", numblocks, blocksize);

	hipEventRecord(start);
   
	if( hipMemcpy(d_x, x, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	for(k = 0; k < iter; k++)
	{
		iloop<<< numblocks, blocksize, N*sizeof(float) >>>(d_A, d_b, N, d_x, d_y); // kernel launch on GPU
	   if( hipMemcpy(d_x, d_y, N*sizeof(float)  , hipMemcpyDeviceToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	}
   if( hipMemcpy(x, d_x, N*sizeof(float)  , hipMemcpyDeviceToHost) != hipSuccess ) printf("CUDA : memory copy error!\n");
	
	//cudaDeviceSynchronize();
	//for(j = 0; j < N; j++) printf("CUDA : %f\n", d_y[j]);

   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
	//Verify : sdiff -s <output> inputs/<input>
   //for(i = 0; i < N; i++)
   //{
   //   for(j = 0; j < N; j++)
   //   {
   //      printf("%d", (int)A[i][j]);
   //      if(j < N-1) printf(" ");
   //   }
   //   printf("\n");
   //}

   //for(i = 0; i < N; i++)
   //{
   //   printf("%d\n", (int)b[i]);
   //}
   //printf("\n\n");

   //for(i = 0; i < N; i++)
   //{
   //   printf("%f\n", x[i]);
   //}
   //printf("\n\n");

	//for(i = 0; i < N; i++) x[i] = d_x[i];

   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
