#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readFile(char* fname, int* N, int* iter, float** A, float** b)
{
   FILE *fp;
   char buf[100];

   int i, j;

   fp = fopen(fname, "r");
   if(!fp)
   {
      *N = 0;
      *iter = 0;
      printf("Stale File Handle\n");
      return;
   }

   if(fscanf(fp, "%s", buf) > 0) *N = atoi(buf);
   if(fscanf(fp, "%s", buf) > 0) *iter = atoi(buf);
   printf("N = %d\nIterations = %d\n", *N, *iter);

   *b = (float*) malloc(*N*sizeof(float));
   *A = (float*) malloc((*N)*(*N)*sizeof(float));
   for(i = 0; i < *N; i++)
   {
      for(j = 0; j < *N; j++)
      {
         fscanf(fp, "%s", buf);
         (*A)[ ((*N*i)+j) ] = (float)atoi(buf);
      }
   }
   fscanf(fp, "%s", buf); // Ignore the "solution" in the text
   for(i = 0; i < *N; i++)
   {
      fscanf(fp, "%s", buf);
      (*b)[i] = (float)atoi(buf);
   }

   fclose(fp);
}




__global__
void iloop(float* A, float* b, int N, float* x, float* y, int offset, int streamsize)
{
	float t;

	int index = offset + (blockIdx.x * blockDim.x) +  threadIdx.x;
	int stride = (gridDim.x * blockDim.x);

	//printf("<<< %d, %d, %d >>>\n", index, (offset+streamsize), stride);

	for(int i = index; i < (offset + streamsize); i += stride)
	{
		t = 0.0;
		for(int j = 0; j < N; j++)
		{
			if(i != j)
			{
				t = t + (( A[ ((N*i)+j) ] ) * x[j]);
			}
		}
		y[i] = ((b[i] - t)/(A[ ((N*i)+i) ]));
	}
}


int main(int argc, char* argv[])
{
   float time = 0.0;
	float maxError = 0.0;
	float* d_A;
	float* d_b;
	float* d_x;
	float* d_y;
	int k;
	int blocksize;
	int numblocks;
	int nstreams;
	int streamsize;
	int devID;

   float* A;
   float* b;
   float* x;
	float* y;
   float* c;
   char* fname;
   int N, iter, i, j, M;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   if(argc >= 2) fname = argv[1];
   else fname = "../inputs/8.txt";
   if(argc >= 3) M = atoi(argv[2]);
   else M = 32;

	//if(argc >= 3) devID = atoi(argv[2]);
	//else devID = 1;

   //cudaDeviceProp prop;
	//if( cudaGetDeviceProperties(&prop, devID) != cudaSuccess) printf("CUDA : Error with Device Properties\n");
	//else printf("Device : %d\n", prop.id);
	//if( cudaSetDevice(devID) != cudaSuccess ) printf("CUDA : Error with Device ID\n");
	
	
	readFile(fname, &N, &iter, &A, &b);

	printf("CUDA : Parsed file %s\n", fname);

	x = (float*) malloc(N*sizeof(float));
	y = (float*) malloc(N*sizeof(float));

	memset(x, 0.0, N*sizeof(float));
	memset(y, 0.0, N*sizeof(float));
	//for(i = 0; i < N; i++)
	//{
	//	x[i] = 0.0;
	//	y[i] = 0.0;
	//}

	if( hipMalloc(&d_A, N*N*sizeof(float)) != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_b, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_x, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");
	if( hipMalloc(&d_y, N*sizeof(float))   != hipSuccess ) printf("CUDA : memory allocation error!\n");

	if( hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_b, b, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	if( hipMemcpy(d_y, y, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");

	nstreams = 2;
	streamsize = N/nstreams;
	blocksize = (int) fmin(M, streamsize);
	numblocks = streamsize/blocksize;
	//numblocks = (N+blocksize-1)/blocksize;
	printf("CUDA : Streams %d, Grid Size %d, Block size %d\n", nstreams, numblocks, blocksize);
 
	hipStream_t streams[nstreams];
   for(i = 0; i < nstreams; i++)
	{
		if( hipStreamCreate(&streams[i]) != hipSuccess ) printf("CUDA : Error with Stream Creation\n");
	}
	
	hipEventRecord(start);
   
	if( hipMemcpy(d_x, x, N*sizeof(float)  , hipMemcpyHostToDevice) != hipSuccess ) printf("CUDA : memory copy error!\n");
	for(k = 0; k < iter/2; k++)
	{
		for(i = 0; i < nstreams; i++)
		{
			int offset = i*streamsize;
			iloop<<< numblocks, blocksize, 0, streams[i] >>>(d_A, d_b, N, d_x, d_y, offset, streamsize); // kernel launch on GPU
			iloop<<< numblocks, blocksize, 0, streams[i] >>>(d_A, d_b, N, d_y, d_x, offset, streamsize); // kernel launch on GPU
		}
	}
   if( hipMemcpy(x, d_x, N*sizeof(float)  , hipMemcpyDeviceToHost) != hipSuccess ) printf("CUDA : memory copy error!\n");
	
   hipEventRecord(stop);
   
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
	
	printf("CUDA : Done Computing Jacobi on GPU\n");
	
   c = (float*) malloc(N*sizeof(float));

   for(i = 0; i < N; i++)
   {
      c[i] = 0;
      for(j = 0; j < N; j++)
      {
         c[i] += A[ ((N*i)+j) ] * x[j];
      }
      //printf("%0.2f\n", c[i]);
		maxError = fmax(maxError, fabs(c[i] - b[i]));
   }

   printf("\nCUDA : Time %f ms\n", time);
	printf("CUDA : MaxError = %f\n\n\n", maxError);

	for(i = 0; i < nstreams; i++)
	{
	   if( hipStreamDestroy(streams[i]) != hipSuccess ) printf("CUDA : Error with Stream Deletion\n");
	}

	hipFree(d_A);
	hipFree(d_b);
	hipFree(d_x);
	hipFree(d_y);
	free(A);
	free(b);
	free(x);
	free(y);
	free(c);

   return 0;
}
